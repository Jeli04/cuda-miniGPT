#include <cstdio>
#include <fstream>
#include <sstream>
#include <string>
#include <iostream>
#include <vector>

#pragma once
#include <hip/hip_runtime.h>


std::vector<std::string> get_qkv_path(int n_blocks, int n_heads, const std::string& folder) {
    std::vector<std::string> paths;
    for (int b = 0; b < n_blocks; ++b) {
        for (int h = 0; h < n_heads; ++h) {
            for (const auto& proj : {"query", "key", "value"}) {
                std::ostringstream oss;
                oss << folder
                    << "block." << b
                    << ".mha.attn_heads." << h
                    << "." << proj << ".weight.txt";
                paths.push_back(oss.str());
            }
        }
    }
    return paths;
}

std::vector<std::string> get_layernorm_paths(
    int n_blocks,
    int ln, 
    const std::string& folder)
{
    std::vector<std::string> paths;
    for (int b = 0; b < n_blocks; ++b) {
        for (const auto& param : {"weight", "bias"}) {
            std::ostringstream oss;
            oss << folder
                << "block." << b
                << ".ln" << ln
                << "." << param << ".txt";
            paths.push_back(oss.str());
        }
    }
    return paths;
}

std::vector<std::string> get_ffwd_paths(
    int n_blocks,
    const std::string& folder)
{
    std::vector<std::string> paths;
    for (int b = 0; b < n_blocks; ++b) {
        for (const auto& layer : {0, 2}) { 
            for (const auto& param : {"bias", "weight"}) {
                std::ostringstream oss;
                oss << folder
                    << "block." << b
                    << ".ffwd." << layer
                    << "." << param << ".txt";
                paths.push_back(oss.str());
            }
        }
    }
    return paths;
}

std::vector<std::string> get_mha_proj_paths(
    int n_blocks,
    const std::string& folder
) {
    std::vector<std::string> paths;
    for (int b = 0; b < n_blocks; ++b) {
        for (const auto& param : {"bias", "weight"}) {
            std::ostringstream oss;
            oss << folder
                << "block." << b
                << ".mha.proj." << param << ".txt";
            paths.push_back(oss.str());
        }
    }
    return paths;
}

std::vector<std::string> get_ln_f_paths(const std::string& folder) {
    std::vector<std::string> paths;
    for (const auto& param : {"bias", "weight"}) {
        std::ostringstream oss;
        oss << folder << "ln_f." << param << ".txt";
        paths.push_back(oss.str());
    }
    return paths;
}

std::vector<std::string> get_lm_head_paths(const std::string& folder) {
    std::vector<std::string> paths;
    for (const auto& param : {"bias", "weight"}) {
        std::ostringstream oss;
        oss << folder << "lm_head." << param << ".txt";
        paths.push_back(oss.str());
    }
    return paths;
}

float* loadMatrix(int rows, int cols, std::string& source){
    float* data = new float[rows * cols]; // or float data[rows * cols];
  
    std::ifstream infile(source);
    if (!infile) {
        std::cerr << "Could not open file.\n";
        exit(1);
    }
  
    std::string line;
    int row = 0;
    while (std::getline(infile, line) && row < rows) {
        if (line.empty()) continue;
        std::istringstream iss(line);
        std::string val;
        int col = 0;
        while (iss >> val && col < cols) {
            data[row * cols + col] = std::stof(val);
            ++col;
        }
        ++row;
    }
  
    for (int i = 0; i < std::min(5, rows * cols); ++i)
        std::cout << data[i] << " ";
    std::cout << std::endl;
    return data;
}

void loadQKVCombined(
    const std::string& source,
    float* dst,
    int rows, 
    int cols
){
    // This function loads the pretrained QKV weights from the disk on host 
    // and combines them into a single matrix for each head.
    // This is a host function, so it will not be run on the device.

    std::ifstream infile(source);
    if (!infile) {
        std::cerr << "Could not open file: " << source << "\n";
        exit(1);
    }
  
    std::string line;
    int row = 0;
    while (std::getline(infile, line) && row < rows) {
        if (line.empty()) continue;
        std::istringstream iss(line);
        std::string val;
        int col = 0;
        while (iss >> val && col < cols) {
            dst[row * cols + col] = std::stof(val);
            ++col;
        }
        ++row;
    }
}


void dumpMatrix(float* matrix, int rows, int cols, const std::string& destination) {
    std::ofstream outfile(destination);
    if (!outfile.is_open()) {
        std::cerr << "Could not open file for writing: " << destination << std::endl;
        return;
    }
    for (int r = 0; r < rows; ++r) {
        for (int c = 0; c < cols; ++c) {
            outfile << matrix[r * cols + c];
            if (c < cols - 1)
                outfile << " ";
        }
        outfile << "\n";
    }
    outfile.close();
}


void printMatrix(float* matrix, int rows, int cols) {
    for (int r = 0; r < rows; ++r) {
        for (int c = 0; c < cols; ++c) {
            // printf("%.4f ", matrix[r * cols + c]);
            printf("Row %d, Col %d: %.4f ", r, c, matrix[r * cols + c]);        
        }
        printf("\n");
    }
}

std::vector<float*> load_qkv_weights(
    int n_blocks, 
    int n_heads, 
    int d_model, 
    int head_dim,
    const std::vector<std::string>& weights_dump
) {
    std::vector<float*> all_weights(n_blocks);

    for(int b = 0; b < n_blocks; b++){
        float* h_W_qkv;
        hipHostAlloc(&h_W_qkv, sizeof(float) * d_model * n_heads * head_dim * 3, hipHostMallocDefault);

        float* h_Q_w = h_W_qkv;
        float* h_K_w = h_W_qkv + head_dim *  n_heads * d_model;
        float* h_V_w = h_W_qkv + 2 * head_dim * n_heads * d_model;

        // load the QKV weights for block b
        for(int i = 0; i < n_heads; i++) {
            int base = 3 * n_heads * b + 3 * i;
            loadQKVCombined(weights_dump[base + 0], h_Q_w + i * head_dim * d_model, head_dim, d_model);
            loadQKVCombined(weights_dump[base + 1], h_K_w + i * head_dim * d_model, head_dim, d_model);
            loadQKVCombined(weights_dump[base + 2], h_V_w + i * head_dim * d_model, head_dim, d_model);
            // printf("%s\n", weights_dump[base + 0].c_str());
            // printf("%s\n", weights_dump[base + 1].c_str());
            // printf("%s\n", weights_dump[base + 2].c_str());
        }

        float* d_W_qkv;
        hipMalloc(&d_W_qkv, sizeof(float) * d_model * n_heads * head_dim * 3);
        hipMemcpy(d_W_qkv, h_W_qkv, sizeof(float) * d_model * n_heads * head_dim * 3, hipMemcpyHostToDevice);
        all_weights[b] = d_W_qkv;
        hipHostFree(h_W_qkv);  // free the host
    }
    return all_weights; // returns a host-side vector of device pointers
}


std::vector<float*> load_layernorm_weights(
    int n_blocks,
    int n_heads,
    int d_model,
    int head_dim,
    const std::vector<std::string>& weights_dump
) {
    std::vector<float*> all_weights(n_blocks);

    for(int b = 0; b < n_blocks; b++){
        std::string gamma_path = weights_dump[2 * b];
        printf("Loading gamma from %s\n", gamma_path.c_str());
        float* h_gamma = loadMatrix(d_model, 1, gamma_path);
        float* d_gamma;
        hipMalloc(&d_gamma, sizeof(float) * n_heads * head_dim * d_model);
        hipMemcpy(d_gamma, h_gamma, sizeof(float) * n_heads * head_dim * d_model, hipMemcpyHostToDevice);
        all_weights[2 * b] = d_gamma;

        std::string beta_path = weights_dump[2 * b + 1];
        printf("Loading beta from %s\n", beta_path.c_str());
        float* h_beta = loadMatrix(d_model, 1, beta_path);
        float* d_beta;
        hipMalloc(&d_beta, sizeof(float) * n_heads * head_dim * d_model);
        hipMemcpy(d_beta, h_beta, sizeof(float) * n_heads * head_dim * d_model, hipMemcpyHostToDevice);
        all_weights[2 * b + 1] = d_beta;
    }

    return all_weights; // returns a host-side vector of device pointers
}


std::vector<float*> load_ffwd_weights(
    int n_blocks,
    int d_model,
    int hidden_dim, 
    const std::vector<std::string>& weights_dump
) {
    std::vector<float*> all_weights(n_blocks * 4);
    for(int b = 0; b < n_blocks; b++){
        // b1
        std::string b1_path = weights_dump[4 * b + 0];
        // printf("Loading b1 from %s\n", b1_path.c_str());
        float* h_b1 = loadMatrix(hidden_dim, 1, b1_path);
        float* d_b1;
        hipMalloc(&d_b1, sizeof(float) * hidden_dim);
        hipMemcpy(d_b1, h_b1, sizeof(float) * hidden_dim, hipMemcpyHostToDevice);
        all_weights[4 * b + 0] = d_b1;

        // w1
        std::string w1_path = weights_dump[4 * b + 1];
        // printf("Loading W1 from %s\n", w1_path.c_str());
        float* h_w1 = loadMatrix(hidden_dim, d_model, w1_path);
        float* d_w1;
        hipMalloc(&d_w1, sizeof(float) * hidden_dim * d_model);
        hipMemcpy(d_w1, h_w1, sizeof(float) * hidden_dim * d_model, hipMemcpyHostToDevice);
        all_weights[4 * b + 1] = d_w1;

        // b2
        std::string b2_path = weights_dump[4 * b + 2];
        // printf("Loading b2 from %s\n", b2_path.c_str());
        float* h_b2 = loadMatrix(d_model, 1, b2_path);
        float* d_b2;
        hipMalloc(&d_b2, sizeof(float) * d_model);
        hipMemcpy(d_b2, h_b2, sizeof(float) * d_model, hipMemcpyHostToDevice);
        all_weights[4 * b + 2] = d_b2;

        // w2
        std::string w2_path = weights_dump[4 * b + 3];
        // printf("Loading W2 from %s\n", w2_path.c_str());
        float* h_w2 = loadMatrix(d_model, hidden_dim, w2_path);
        float* d_w2;
        hipMalloc(&d_w2, sizeof(float) * d_model * hidden_dim);
        hipMemcpy(d_w2, h_w2, sizeof(float) * d_model * hidden_dim, hipMemcpyHostToDevice);
        all_weights[4 * b + 3] = d_w2;
    }

    return all_weights; // Host vector of device pointers
}


std::vector<float*> load_mha_proj_weights(
    int n_blocks,
    int d_model,
    const std::vector<std::string>& weights_dump
) {
    std::vector<float*> all_weights(n_blocks * 2);
    for(int b = 0; b < n_blocks; ++b) {
        // Bias
        std::string bias_path = weights_dump[2 * b + 0];
        float* h_bias = loadMatrix(d_model, 1, bias_path);
        float* d_bias;
        hipMalloc(&d_bias, sizeof(float) * d_model);
        hipMemcpy(d_bias, h_bias, sizeof(float) * d_model, hipMemcpyHostToDevice);
        all_weights[2 * b + 0] = d_bias;

        // Weight
        std::string weight_path = weights_dump[2 * b + 1];
        float* h_weight = loadMatrix(d_model, d_model, weight_path); // assuming weight is [d_model, d_model]
        float* d_weight;
        hipMalloc(&d_weight, sizeof(float) * d_model * d_model);
        hipMemcpy(d_weight, h_weight, sizeof(float) * d_model * d_model, hipMemcpyHostToDevice);
        all_weights[2 * b + 1] = d_weight;
    }
    return all_weights;
}

std::vector<float*> load_ln_f_weights(
    int d_model,
    const std::vector<std::string>& weights_dump
) {
    std::vector<float*> all_weights(2); 

    // Bias
    std::string bias_path = weights_dump[0];
    float* h_bias = loadMatrix(d_model, 1, bias_path);
    float* d_bias;
    hipMalloc(&d_bias, sizeof(float) * d_model);
    hipMemcpy(d_bias, h_bias, sizeof(float) * d_model, hipMemcpyHostToDevice);
    all_weights[0] = d_bias;

    // Weight
    std::string weight_path = weights_dump[1];
    float* h_weight = loadMatrix(d_model, 1, weight_path);
    float* d_weight;
    hipMalloc(&d_weight, sizeof(float) * d_model);
    hipMemcpy(d_weight, h_weight, sizeof(float) * d_model, hipMemcpyHostToDevice);
    all_weights[1] = d_weight;

    return all_weights;
}

std::vector<float*> load_lm_head_weights(
    int vocab_size,
    int d_model,
    const std::vector<std::string>& weights_dump
) {
    std::vector<float*> all_weights(2); // bias and weight

    // Bias
    std::string bias_path = weights_dump[0];
    float* h_bias = loadMatrix(vocab_size, 1, bias_path);
    float* d_bias;
    hipMalloc(&d_bias, sizeof(float) * vocab_size);
    hipMemcpy(d_bias, h_bias, sizeof(float) * vocab_size, hipMemcpyHostToDevice);
    all_weights[0] = d_bias;

    // Weight
    std::string weight_path = weights_dump[1];
    float* h_weight = loadMatrix(vocab_size, d_model, weight_path);
    float* d_weight;
    hipMalloc(&d_weight, sizeof(float) * vocab_size * d_model);
    hipMemcpy(d_weight, h_weight, sizeof(float) * vocab_size * d_model, hipMemcpyHostToDevice);
    all_weights[1] = d_weight;

    return all_weights;
}