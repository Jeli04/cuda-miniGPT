#include <stdio.h>
#include <iostream>

#define TILE_SIZE 16
#pragma once
#include <hip/hip_runtime.h>



__global__ void mysgemm(int m, int n, int k, bool A_t, bool B_t, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     * Use shared memory for tiling
     *
     ********************************************************************/

    /*************************************************************************/
    // INSERT KERNEL CODE HERE
    __shared__ float As[TILE_SIZE][TILE_SIZE]; // shared memory for A
    __shared__ float Bs[TILE_SIZE][TILE_SIZE]; // shared memory for B

    int row, col;
    const float *A_ptr;
    const float *B_ptr;
    float *C_ptr;

    // if we have a 3D grid, we can use the z dimension to handle multiple matrices
    if (gridDim.z > 1) {
        row = blockIdx.y * blockDim.y + threadIdx.y;   // row tile
        col = blockIdx.z * blockDim.x + threadIdx.x;   // col tile
        
        // x here is the first dimension of the grid 
        A_ptr = A + blockIdx.x * m * k; // pointer to the A matrix for this block
        B_ptr = B + blockIdx.x * k * n; // pointer to the B matrix for this block
        C_ptr = C + blockIdx.x * m * n; // pointer to the C matrix for this block
    }
    else {
        row = blockIdx.y * blockDim.y + threadIdx.y; 
        col = blockIdx.x * blockDim.x + threadIdx.x; 

        A_ptr = A;
        B_ptr = B;
        C_ptr = C;
    }

    float Cvalue = 0.0f;
    // iterate across the tiles 
    for(int tile_idx = 0; tile_idx < (k + TILE_SIZE - 1) / TILE_SIZE; tile_idx++) {
        int A_row = A_t ? tile_idx * TILE_SIZE + threadIdx.x : row;
        int A_col = A_t ? row : tile_idx * TILE_SIZE + threadIdx.x;
        if (A_row < (A_t ? k : m) && A_col < (A_t ? m : k)){
            As[threadIdx.y][threadIdx.x] = A_ptr[(A_row * k) + A_col];
        }
        else{
            As[threadIdx.y][threadIdx.x] = 0.0f;
        }

        int B_stride = B_t ? k : n; // stride for B based on whether B is transposed or not
        int B_col = B_t ? tile_idx * TILE_SIZE + threadIdx.y : col; 
        int B_row = B_t ? col : tile_idx * TILE_SIZE + threadIdx.y;
        if (B_row < (B_t ? n : k) && B_col < (B_t ? k : n)){
            Bs[threadIdx.y][threadIdx.x] = B_ptr[B_row * B_stride + B_col];  // * stide (n or k) here since B is column major
        }
        else{
            Bs[threadIdx.y][threadIdx.x] = 0.0f;
        }

        // synchronize to make sure the data is loaded
        __syncthreads();

        // dot product of the column and row in the tile
        for(int i = 0; i < TILE_SIZE; i++) {
            // compute the value of C
            Cvalue += As[threadIdx.y][i] * Bs[i][threadIdx.x];
        }
        __syncthreads();

    }

    if (row < m && col < n){
        C_ptr[row*n + col] = Cvalue;
    }
    /*************************************************************************/
}

void basicSgemm(int m, int n, int k, bool A_t, bool B_t, const float *A, const float *B, float *C)
{
    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = TILE_SIZE;
	
    /*************************************************************************/
    //INSERT CODE HERE
    dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE); // create the block dim 
    // we have n as first dimension and m as second dimension so easier to access for x and y (row and col)
    dim3 dim_grid((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE); // create the grid dim

    /*************************************************************************/

    // Invoke CUDA kernel -----------------------------------------------------

    /*************************************************************************/
    //INSERT CODE HERE
    mysgemm<<<dim_grid, dim_block>>>(m, n, k, A_t, B_t, A, B, C);

    /*************************************************************************/
}

