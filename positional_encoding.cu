#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "tools.h"
#include "sgemm.h"
#include "positional_encoding.h"

__global__ void add_embeddings(const float* token_emb, const float* pos_emb, float* output, int seq_len, int d_model) {
    int i = blockIdx.y * blockDim.y + threadIdx.y; // sequence position
    int j = blockIdx.x * blockDim.x + threadIdx.x; // embedding dimension
    
    if (i < seq_len && j < d_model) {
        output[i * d_model + j] = token_emb[i * d_model + j] + pos_emb[i * d_model + j];
    }
}

void embed_sequence_sgemm(
    float* d_output,           
    const float* d_token_table, 
    const float* d_pos_table,  
    const int* h_token_sequence,
    int seq_len,
    int d_model,
    int vocab_size,
    int max_seq_len,
    PositionalEncodingResources* resources 
) {
    hipMemset(resources->d_token_onehot, 0, seq_len * vocab_size * sizeof(float));
    hipMemset(resources->d_pos_onehot, 0, seq_len * max_seq_len * sizeof(float));
    
    memset(resources->h_token_onehot, 0, seq_len * vocab_size * sizeof(float));
    memset(resources->h_pos_onehot, 0, seq_len * max_seq_len * sizeof(float));
    
    for (int i = 0; i < seq_len; i++) {
        resources->h_token_onehot[i * vocab_size + h_token_sequence[i]] = 1.0f;
        resources->h_pos_onehot[i * max_seq_len + i] = 1.0f; 
    }

    hipMemcpy(resources->d_token_onehot, resources->h_token_onehot, 
               seq_len * vocab_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(resources->d_pos_onehot, resources->h_pos_onehot, 
               seq_len * max_seq_len * sizeof(float), hipMemcpyHostToDevice);
    
    basicSgemm(seq_len, d_model, vocab_size, false, false, 
               resources->d_token_onehot, d_token_table, resources->d_token_embeddings);
    
    basicSgemm(seq_len, d_model, max_seq_len, false, false, 
               resources->d_pos_onehot, d_pos_table, resources->d_pos_embeddings);
    
    dim3 block(16, 16);
    dim3 grid((d_model + block.x - 1) / block.x, (seq_len + block.y - 1) / block.y);
    add_embeddings<<<grid, block>>>(resources->d_token_embeddings, resources->d_pos_embeddings, 
                                   d_output, seq_len, d_model);
    hipDeviceSynchronize();
}


void test_positional_encoding_sgemm() {
    printf("=== POSITIONAL ENCODING WITH SGEMM ===\n");
    
    int vocab_size = 84;
    int d_model = 128;
    int max_seq_len = 64;
    int seq_len = 18;
    
    PositionalEncodingResources pos_resources;
    initialize_positional_encoding_resources(&pos_resources, max_seq_len, vocab_size, d_model);
    
    std::string weights_folder = "./weights_dump/";
    std::string token_file = weights_folder + "token_embedding_table.weight.txt";
    std::string pos_file = weights_folder + "position_embedding_table.weight.txt";
    
    float* h_token_table = loadMatrix(vocab_size, d_model, token_file);
    float* h_pos_table = loadMatrix(max_seq_len, d_model, pos_file);
    
    printf("Loaded token embedding table: %d x %d\n", vocab_size, d_model);
    printf("Loaded position embedding table: %d x %d\n", max_seq_len, d_model);
    
    float* d_token_table;
    float* d_pos_table;
    float* d_output;
    
    hipMalloc(&d_token_table, vocab_size * d_model * sizeof(float));
    hipMalloc(&d_pos_table, max_seq_len * d_model * sizeof(float));
    hipMalloc(&d_output, seq_len * d_model * sizeof(float));
    
    hipMemcpy(d_token_table, h_token_table, vocab_size * d_model * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_pos_table, h_pos_table, max_seq_len * d_model * sizeof(float), hipMemcpyHostToDevice);
    
    int h_tokens[18] = {45, 70, 1, 57, 60, 1, 70, 73, 1, 69, 70, 75, 1, 75, 70, 1, 57, 60};
    
    printf("Processing sequence using SGEMM matrix multiplication...\n");
    
    embed_sequence_sgemm(
        d_output,
        d_token_table,
        d_pos_table,
        h_tokens,
        seq_len,
        d_model,
        vocab_size,
        max_seq_len,
        &pos_resources
    );
    
    float* h_result = (float*)malloc(seq_len * d_model * sizeof(float));
    hipMemcpy(h_result, d_output, seq_len * d_model * sizeof(float), hipMemcpyDeviceToHost);
    
    printf("Sequence embedding (first token, first 8 dims): ");
    for (int i = 0; i < 8; i++) {
        printf("%.6f ", h_result[i]);
    }
    printf("\n");
    
    dumpMatrix(h_result, seq_len, d_model, "./positional_embedding_result.txt");
    printf("Saved positional results to: positional_embedding_result.txt\n");
    
    cleanup_positional_encoding_resources(&pos_resources);
    hipFree(d_token_table);
    hipFree(d_pos_table);
    hipFree(d_output);
    free(h_token_table);
    free(h_pos_table);
    free(h_result);
}
