#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hipblas.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include <sys/time.h>
#include <time.h>
#include "generate.h"
#include "minigpt.h"
#include "softmax.h"
#include "tools.h"

#define VOCAB_SIZE 84

double get_wall_time() {
    struct timeval time;
    if (gettimeofday(&time, NULL)) {
        return 0;
    }
    return (double)time.tv_sec + (double)time.tv_usec * .000001;
}

// Setup random states
__global__ void setup_random_states(hiprandState* states, unsigned long seed, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        hiprand_init(seed, idx, 0, &states[idx]);
    }
}

__global__ void multinomial_sample_kernel(
    const float* probs,
    int* selected_token,
    hiprandState* states,
    int vocab_size
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx != 0) return;

    hiprandState* state = &states[idx];
    float coin = hiprand_uniform(state);

    float cumulative_prob = 0.0f;
    int selected = 0;
    
    for(int i = 0; i < vocab_size; i++){
        cumulative_prob += probs[i];
        if(coin <= cumulative_prob){
            selected = i;
            break;
        }
    }

    if (coin > cumulative_prob) {
        selected = vocab_size - 1;
    }

    *selected_token = selected;
}

int* text_to_tokens(char** vocab, int vocab_size, const char* text, int* num_tokens) {
    int text_len = strlen(text);
    int* token_ids = (int*)malloc(text_len * sizeof(int));
    int valid_tokens = 0;
    
    for (int i = 0; i < text_len; i++) {
        char target_char = text[i];
        int token_id = -1;
        
        // Find matching token
        for (int j = 0; j < vocab_size; j++) {
            if (vocab[j] && strlen(vocab[j]) == 1 && vocab[j][0] == target_char) {
                token_id = j;
                break;
            }
        }
        
        if (token_id >= 0) {
            token_ids[valid_tokens] = token_id;
        } else {
            // Default to space token
            for (int j = 0; j < vocab_size; j++) {
                if (vocab[j] && strlen(vocab[j]) == 1 && vocab[j][0] == ' ') {
                    token_ids[valid_tokens] = j;
                    break;
                }
            }
        }
        valid_tokens++;
    }
    
    *num_tokens = valid_tokens;
    return token_ids;
}

// void debug_conditioning(int* input_tokens, int input_length, 
//     int* conditioned_tokens, int conditioned_length, 
//     int block_size, char** vocab, int vocab_size) {
// printf("\n=== CONDITIONING DEBUG ===\n");
// printf("Input length: %d, Block size: %d\n", input_length, block_size);

// printf("Original input tokens: ");
// for (int i = 0; i < input_length; i++) {
// printf("%d ", input_tokens[i]);
// }
// printf("\n");

// printf("Original input text: '");
// for (int i = 0; i < input_length; i++) {
// if (input_tokens[i] >= 0 && input_tokens[i] < vocab_size && vocab[input_tokens[i]]) {
// printf("%s", vocab[input_tokens[i]]);
// }
// }
// printf("'\n");

// printf("Conditioned tokens (%d): ", conditioned_length);
// for (int i = 0; i < conditioned_length; i++) {
// printf("%d ", conditioned_tokens[i]);
// }
// printf("\n");

// printf("Conditioned text: '");
// for (int i = 0; i < conditioned_length; i++) {
// if (conditioned_tokens[i] >= 0 && conditioned_tokens[i] < vocab_size && vocab[conditioned_tokens[i]]) {
// printf("%s", vocab[conditioned_tokens[i]]);
// }
// }
// printf("'\n");

// if (input_length > block_size) {
// int start_idx = input_length - block_size;
// printf("CONDITIONING APPLIED: Kept last %d tokens (skipped first %d)\n", 
// block_size, start_idx);
// printf("Skipped text: '");
// for (int i = 0; i < start_idx; i++) {
// if (input_tokens[i] >= 0 && input_tokens[i] < vocab_size && vocab[input_tokens[i]]) {
// printf("%s", vocab[input_tokens[i]]);
// }
// }
// printf("'\n");
// } else {
// printf("NO CONDITIONING NEEDED: Input fits within block_size\n");
// }
// printf("========================\n\n");
// }


void generate_tokens_contextual(
    int block_size,
    int d_model,
    int n_heads,
    int head_dim,
    int n_blocks,
    int* input_tokens,
    int input_length,
    int max_new_tokens,
    int vocab_size,
    char** vocab,
    hiprandState* d_states,
    PositionalEncodingResources& pos_resources,
    MiniGPT& gpt_model
) {
    printf("\n===TOKEN GENERATION WITH CONDITIONING CHECK ===\n");
    printf("here");
    double start_time = get_wall_time();
    printf("here");

    float* d_logits;
    hipMalloc(&d_logits, block_size * vocab_size * sizeof(float));
    float* d_probs;
    hipMalloc(&d_probs, vocab_size * sizeof(float));

    printf("here");

    int* conditioned_tokens;
    int conditioned_length;
    char* full_text = (char*)malloc(10000);
    strcpy(full_text, "");
    int* token_sequence;


    // Generate tokens one by one
    for (int step = 0; step < max_new_tokens; step++) {
        // Allocate GPU buffer for input
        float* d_input;
        hipMalloc(&d_input, block_size * d_model * sizeof(float));

        // intialize selected token
        int* d_selected_token;
        hipMalloc(&d_selected_token, sizeof(int));

        // crops the context to the last block_size tokens [:, -block_size:]
        if (input_length > block_size) {
            conditioned_length = block_size;
            conditioned_tokens = (int*)malloc(conditioned_length * sizeof(int));
    
            int start_idx = input_length - block_size;
            
            memcpy(conditioned_tokens, input_tokens + start_idx, conditioned_length * sizeof(int));
        } 
        else {
            conditioned_length = input_length;
            conditioned_tokens = (int*)malloc(conditioned_length * sizeof(int));
        
            memcpy(conditioned_tokens, input_tokens, conditioned_length * sizeof(int));
        }
  
        int max_seq_len = conditioned_length + max_new_tokens;
        token_sequence = (int*)malloc(max_seq_len * sizeof(int));
        
        memcpy(token_sequence, conditioned_tokens, conditioned_length * sizeof(int));
        int current_length = conditioned_length;
        
        for (int i = 0; i < conditioned_length; i++) {
            if (conditioned_tokens[i] >= 0 && conditioned_tokens[i] < vocab_size && vocab[conditioned_tokens[i]]) {
                strcat(full_text, vocab[conditioned_tokens[i]]);
            }
        }    

        // intialize default logits
        float* logits = (float*)malloc(block_size * vocab_size * sizeof(float));
        for (int i = 0; i < block_size * vocab_size; i++) {
            logits[i] = 1.0f; 
        }
        // Copy logits to device
        hipMemcpy(d_logits, logits, block_size * vocab_size * sizeof(float), hipMemcpyHostToDevice);

        // forward call from transformer
        gpt_model.forward_pass(
            block_size, // seq_len
            max_seq_len,
            token_sequence,
            pos_resources,
            d_input,
            d_logits,   // d_output
            block_size,
            n_heads,
            d_model,
            head_dim,
            n_blocks,
            vocab_size
        );

        // Apply softmax to get probabilities
        float* d_probs;
        hipMalloc(&d_probs, vocab_size * sizeof(float));
        softmax(d_logits, d_probs, 1, vocab_size);

        multinomial_sample_kernel<<<1, 1>>>(d_probs, d_selected_token, d_states, vocab_size);
        hipDeviceSynchronize();

        int next_token;
        hipMemcpy(&next_token, d_selected_token, sizeof(int), hipMemcpyDeviceToHost);
        
        if (next_token < 0 || next_token >= vocab_size) {
            free(logits);
            break;
        }

        token_sequence[current_length] = next_token;
        current_length++;
        
        if (vocab[next_token]) {
            strcat(full_text, vocab[next_token]);
        }
        
        free(logits);
    }
    
    double end_time = get_wall_time();
    double total_time = end_time - start_time;
    
    printf("\n=== GENERATED TEXT ===\n");
    printf("'%s'\n", full_text);
    printf("======================\n");
    printf("Generation time: %.3f ms\n", total_time * 1000);
    
    hipFree(d_probs);
    free(conditioned_tokens);
    free(token_sequence);
    free(full_text);
}

// char** load_vocab_json(const char* filename, int* vocab_size) {
//     *vocab_size = 128; // Increase to cover full ASCII range
//     char** vocab = (char**)malloc(*vocab_size * sizeof(char*));
    
//     // Create ASCII character vocabulary (0-127)
//     for (int i = 0; i < *vocab_size; i++) {
//         vocab[i] = (char*)malloc(2 * sizeof(char));
//         vocab[i][0] = (char)i;
//         vocab[i][1] = '\0';
//     }
    
//     return vocab;
// }


// int main() {
    // hipblasHandle_t cublasHandle;
    // hipblasCreate(&cublasHandle);
    
    // int vocab_size;
    // char** vocab = load_vocab_json("vocab.json", &vocab_size);
    // if (!vocab) {
    //     printf("Failed to load vocabulary\n");
    //     return 1;
    // }
    
    // printf("=== TRANSFORMER VALIDATION TEST ===\n");
    // printf("Loaded vocabulary: %d tokens\n", vocab_size);
    
    // // Test configuration - match your Python model exactly
    // TransformerBlockCofig config;
    // config.block_size = 32;
    // config.n_heads = 4;
    // config.d_model = 102;
    // config.head_dim = config.d_model / config.n_heads;
    // config.n_blocks = 1;
    // config.vocab_size = vocab_size;
    
    // printf("Config: block_size=%d, n_heads=%d, d_model=%d, head_dim=%d\n", 
    //        config.block_size, config.n_heads, config.d_model, config.head_dim);
    
    // // Allocate device memory
    // float *d_input, *d_output, *d_residual;
    // float *d_logits, *d_probs;
    // int *d_selected_token;
    // hiprandState *d_states;
    
    // hipMalloc(&d_input, config.block_size * config.d_model * sizeof(float));
    // hipMalloc(&d_output, config.block_size * config.d_model * sizeof(float));
    // hipMalloc(&d_residual, config.block_size * config.d_model * sizeof(float));
    // hipMalloc(&d_logits, config.vocab_size * sizeof(float));
    // hipMalloc(&d_probs, config.vocab_size * sizeof(float));
    // hipMalloc(&d_selected_token, sizeof(int));
    // hipMalloc(&d_states, config.vocab_size * sizeof(hiprandState));
    
    // unsigned long fixed_seed = 12345;
    // setup_random_states<<<(config.vocab_size + 255) / 256, 256>>>(d_states, fixed_seed, config.vocab_size);
    // hipDeviceSynchronize();
    
    // // Multiple test cases
    // const char* test_inputs[] = {
    //     "Hi",                                          // Short (2 chars < 32)
    //     "Hello world",                                // Medium (11 chars < 32)  
    //     "This is a longer test sentence that should definitely exceed the block size limit of thirty-two characters and trigger conditioning",  // Long (>32 chars)
    //     "A",                                          // Very short (1 char)
    //     "The quick brown fox jumps over the lazy dog and then continues running for a very long time"  // Very long (>32 chars)
    // };
    // int num_tests = sizeof(test_inputs) / sizeof(test_inputs[0]);
    
    // for (int test_idx = 0; test_idx < num_tests; test_idx++) {
    //     printf("\n=== TEST CASE %d ===\n", test_idx + 1);
    //     const char* input_text = test_inputs[test_idx];
    //     int input_length;
    //     int* input_tokens = text_to_tokens(vocab, vocab_size, input_text, &input_length);
        
    //     printf("Input: '%s' -> %d tokens: ", input_text, input_length);
    //     for (int i = 0; i < input_length; i++) {
    //         printf("%d ", input_tokens[i]);
    //     }
    //     printf("\n");
        
    //     // Reset random state for each test
    //     setup_random_states<<<(config.vocab_size + 255) / 256, 256>>>(d_states, fixed_seed + test_idx, config.vocab_size);
    //     hipDeviceSynchronize();
        
    //     // Generate with CUDA
    //     generate_tokens_contextual(
    //         input_tokens,
    //         input_length,
    //         5,  
    //         vocab_size,
    //         vocab,
    //         d_output,
    //         d_logits,
    //         d_selected_token,
    //         d_states,
    //         config
    //     );
        
    //     free(input_tokens);
    // }
    
    // // Cleanup
    // hipFree(d_input);
    // hipFree(d_output);
    // hipFree(d_residual);
    // hipFree(d_logits);
    // hipFree(d_probs);
    // hipFree(d_selected_token);
    // hipFree(d_states);
    
    // for (int i = 0; i < vocab_size; i++) {
    //     if (vocab[i]) {
    //         free(vocab[i]);
    //     }
    // }
    // free(vocab);
    
    // hipblasDestroy(cublasHandle);
    
    // printf("\n=== VALIDATION COMPLETE ===\n");
//     return 0;
// }
